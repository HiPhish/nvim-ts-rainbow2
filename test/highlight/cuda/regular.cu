
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cstdio>

/* A function declaration */
int add(int, int);

// Structure and class definitions
struct Point2D {
public:
	int x;
	int y;
};

class Point3D {
public:
	int x;
	int y;
	int z;
};

/* A function definition */
int add(int x, int y) {
	if (!y) {
		if (1) {
			if (1) {
				if (1) {
					return x;
				}
			}
		}
	}

	while (0) {
		while (0) {
			while (0) {
				;
			}
		}
	}

	for (int i = 0; i < 0; i++) {
		for (int j = 0; j < 0; j++) {
			for (int k = 0; k < 0; k++) {
				;
			}
		}
	}

	return add(x + 1, y - 1);
}

template <typename T> T myMax(T x, T y) {
	return (x > y) ? x : y;
}

float int2float(int i) {
	return (float)i;
}

void do_nothing_with_vector(std::vector<std::vector<std::vector<int>>> v) {
	return;
}

__global__ void add_array(int *a, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        a[i] += 1;
    }
}

void call_device() {
	int *dev_a;
	hipMalloc(&dev_a, 10 * sizeof(int));
	add_array<<<1, 10, 1>>>(dev_a, 10);
	hipFree(dev_a);
}

int main(int argc, char *argv[]) {
	auto a {10};
	auto b (5);
	auto result = add(a, b);
	printf("The sum of %d and %d is %d", ((((a)))), b, result);
	int indices[] = {0, };
	auto i = indices[indices[indices[indices[indices[indices[0]]]]]];
	return 0;
}
